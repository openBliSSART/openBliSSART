
#include <hip/hip_runtime.h>
//
// This file is part of openBliSSART.
//
// Copyright (c) 2007-2011, Alexander Lehmann <lehmanna@in.tum.de>
//                          Felix Weninger <felix@weninger.de>
//                          Bjoern Schuller <schuller@tum.de>
//
// Institute for Human-Machine Communication
// Technische Universitaet Muenchen (TUM), D-80333 Munich, Germany
//
// openBliSSART is free software: you can redistribute it and/or modify it under
// the terms of the GNU General Public License as published by the Free Software
// Foundation, either version 2 of the License, or (at your option) any later
// version.
//
// openBliSSART is distributed in the hope that it will be useful, but WITHOUT
// ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
// details.
//
// You should have received a copy of the GNU General Public License along with
// openBliSSART.  If not, see <http://www.gnu.org/licenses/>.
//
#undef HAVE_CUDA
#ifdef HAVE_CUDA

#include <cuda.h>
#include <blissart/linalg/common.h>


namespace blissart {


namespace linalg {


namespace gpu {


int blocksize = 4;


__global__ void MatrixAdd_d(const Elem *a, const Elem *b, Elem *c, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col * rows + row;
    if(col < cols && row < rows) 
        c[index] = a[index] + b[index];
}


void apply_add(const Elem* a, const Elem* b, Elem* c, int m, int n) {
    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(n / dimBlock.x + 1, m / dimBlock.y + 1);
    MatrixAdd_d<<<dimGrid, dimBlock>>>(a, b, c, m, n);
    cudaThreadSynchronize();
}


__global__ void MatrixSub_d(const Elem *a, const Elem *b, Elem *c, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col * rows + row;
    if(col < cols && row < rows) 
        c[index] = a[index] - b[index];
}


void apply_sub(const Elem* a, const Elem* b, Elem* c, int m, int n) {
    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(n / dimBlock.x + 1, m / dimBlock.y + 1);
    MatrixSub_d<<<dimGrid, dimBlock>>>(a, b, c, m, n);
    cudaThreadSynchronize();
}


__global__ void MatrixMul_d(const Elem *a, const Elem *b, Elem *c, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col * rows + row;
    if(col < cols && row < rows) 
        c[index] = a[index] * b[index];
}


void apply_mul(const Elem* a, const Elem* b, Elem* c, int m, int n) {
    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(n / dimBlock.x + 1, m / dimBlock.y + 1);
    MatrixMul_d<<<dimGrid, dimBlock>>>(a, b, c, m, n);
    cudaThreadSynchronize();
}


__global__ void MatrixDiv_d(const Elem *a, const Elem *b, Elem *c, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col * rows + row;
    if(col < cols && row < rows) 
        c[index] = a[index] / b[index];
}


void apply_div(const Elem* a, const Elem* b, Elem* c, int m, int n) {
    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(n / dimBlock.x + 1, m / dimBlock.y + 1);
    MatrixDiv_d<<<dimGrid, dimBlock>>>(a, b, c, m, n);
    cudaThreadSynchronize();
}


__global__ void MatrixPow_d(const Elem *a, const Elem b, Elem *c, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col * rows + row;
    if(col < cols && row < rows) 
        c[index] = pow(a[index], b);
}


void apply_pow(const Elem* a, const Elem b, Elem* c, int m, int n) {
    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(n / dimBlock.x + 1, m / dimBlock.y + 1);
    MatrixPow_d<<<dimGrid, dimBlock>>>(a, b, c, m, n);
    cudaThreadSynchronize();
}


__global__ void SetZero_d(Elem* a, int rows, int cols, 
                          int startRow, int startCol, int endRow, int endCol)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col * rows + row;
    if (col >= startCol && col <= endCol && row >= startRow && row <= endRow)
        a[index] = 0.0f;
}


void set_to_zero(Elem* a, int rows, int cols, 
                 int startRow, int startCol, int endRow, int endCol)
{
    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(cols / dimBlock.x + 1, rows / dimBlock.y + 1);
    SetZero_d<<<dimGrid, dimBlock>>>
        (a, rows, cols, startRow, startCol, endRow, endCol);
    cudaThreadSynchronize();
}


__global__ void Floor_d(Elem* a, const Elem floor, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col * rows + row;
    if(col < cols && row < rows && a[index] < floor) 
        a[index] = floor;
}


void apply_floor(Elem* a, const Elem floor, int rows, int cols)
{
    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(cols / dimBlock.x + 1, rows / dimBlock.y + 1);
    Floor_d<<<dimGrid, dimBlock>>>(a, floor, rows, cols);
    cudaThreadSynchronize();
}


} // namespace gpu


} // namespace linalg


} // namespace blissart
#endif
